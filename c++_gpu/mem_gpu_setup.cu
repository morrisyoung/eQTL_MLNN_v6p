#include "hip/hip_runtime.h"
// Utilities and system includes
#include <assert.h>
#include <helper_string.h>  // helper for shared functions common to CUDA Samples
#include <sys/time.h>
#include <time.h>       /* clock_t, clock, CLOCKS_PER_SEC */
#include <random>
#include <chrono>		/* sys time */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <vector>


// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// private
#include "global.h"
#include "library.h"
#include "mem_gpu_setup.h"




using namespace std;




void mem_gpu_init()
{
	//int dimension1, dimension2;
	int dimension2;


	//== d_X_batch, d_Z_batch, d_Y_batch, d_Y_batch_exp, d_cellfactor_batch, d_cellfactor_batch_new
	//
	dimension2 = X.get_dimension2();
	checkCudaErrors(hipMalloc((void **) &d_X_batch, (size_batch*dimension2)*sizeof(float)));
	//
	dimension2 = Z.get_dimension2();
	checkCudaErrors(hipMalloc((void **) &d_Z_batch, (size_batch*dimension2)*sizeof(float)));
	//
	dimension2 = Y.get_dimension3();
	checkCudaErrors(hipMalloc((void **) &d_Y_batch, (size_batch*dimension2)*sizeof(float)));
	checkCudaErrors(hipMalloc((void **) &d_Y_batch_exp, (size_batch*dimension2)*sizeof(float)));
	checkCudaErrors(hipMalloc((void **) &d_error_batch, (size_batch*dimension2)*sizeof(float)));
	//
	checkCudaErrors(hipMalloc((void **) &d_cellfactor_batch, (size_batch*D)*sizeof(float)));
	//
	checkCudaErrors(hipMalloc((void **) &d_cellfactor_batch_new, (size_batch*(D+1))*sizeof(float)));




	//== list_cis_start, d_list_cis_start
	int * list_cis_start = mapping_cis.get_list_start();
	checkCudaErrors(hipMalloc((void **) &d_list_cis_start, J*sizeof(int)));
	checkCudaErrors(hipMemcpy(d_list_cis_start, list_cis_start, J*sizeof(int), hipMemcpyHostToDevice));
	//== list_cis_end, d_list_cis_end
	int * list_cis_end = mapping_cis.get_list_end();
	checkCudaErrors(hipMalloc((void **) &d_list_cis_end, J*sizeof(int)));
	checkCudaErrors(hipMemcpy(d_list_cis_end, list_cis_end, J*sizeof(int), hipMemcpyHostToDevice));
	//== list_beta_cis_start, d_list_beta_cis_start
	int * list_beta_cis_start = beta_cis.get_list_start();
	checkCudaErrors(hipMalloc((void **) &d_list_beta_cis_start, J*sizeof(int)));
	checkCudaErrors(hipMemcpy(d_list_beta_cis_start, list_beta_cis_start, J*sizeof(int), hipMemcpyHostToDevice));
	//== d_list_beta_cis_geneindex
	int * list_beta_cis_geneindex = beta_cis.get_list_beta_cis_geneindex();
	int amount = beta_cis.get_amount();
	checkCudaErrors(hipMalloc((void **) &d_list_beta_cis_geneindex, amount*sizeof(int)));
	checkCudaErrors(hipMemcpy(d_list_beta_cis_geneindex, list_beta_cis_geneindex, amount*sizeof(int), hipMemcpyHostToDevice));
	//== d_beta_cis_sub, d_der_cis_sub
	//int amount = beta_cis.get_amount();
	checkCudaErrors(hipMalloc((void **) &d_beta_cis_sub, amount*sizeof(float)));
	checkCudaErrors(hipMalloc((void **) &d_der_cis_sub, amount*sizeof(float)));


	//== d_beta_batch, d_beta_batch_reshape, d_der_batch
	int dimension1_beta_batch = beta_batch.get_dimension1();
	int dimension2_beta_batch = beta_batch.get_dimension2();
	checkCudaErrors(hipMalloc((void **) &d_beta_batch, (dimension1_beta_batch*dimension2_beta_batch)*sizeof(float)));
	checkCudaErrors(hipMalloc((void **) &d_beta_batch_reshape, (dimension1_beta_batch*dimension2_beta_batch)*sizeof(float)));
	checkCudaErrors(hipMalloc((void **) &d_der_batch, (dimension1_beta_batch*dimension2_beta_batch)*sizeof(float)));

	//== d_beta_cellfactor1, d_beta_cellfactor1_reshape, d_der_cellfactor1
	int dimension1_beta_cellfactor1 = beta_cellfactor1.get_dimension1();
	int dimension2_beta_cellfactor1 = beta_cellfactor1.get_dimension2();
	checkCudaErrors(hipMalloc((void **) &d_beta_cellfactor1, (dimension1_beta_cellfactor1*dimension2_beta_cellfactor1)*sizeof(float)));
	checkCudaErrors(hipMalloc((void **) &d_beta_cellfactor1_reshape, (dimension1_beta_cellfactor1*dimension2_beta_cellfactor1)*sizeof(float)));
	checkCudaErrors(hipMalloc((void **) &d_der_cellfactor1, (dimension1_beta_cellfactor1*dimension2_beta_cellfactor1)*sizeof(float)));

	//== d_beta_cellfactor2_sub, d_beta_cellfactor2_sub_reshape, d_der_cellfactor2_sub
	int dimension1_beta_cellfactor2 = beta_cellfactor2.get_dimension2();
	int dimension2_beta_cellfactor2 = beta_cellfactor2.get_dimension3();
	checkCudaErrors(hipMalloc((void **) &d_beta_cellfactor2_sub, (dimension1_beta_cellfactor2*dimension2_beta_cellfactor2)*sizeof(float)));
	checkCudaErrors(hipMalloc((void **) &d_beta_cellfactor2_sub_reshape, (dimension1_beta_cellfactor2*dimension2_beta_cellfactor2)*sizeof(float)));
	checkCudaErrors(hipMalloc((void **) &d_der_cellfactor2_sub, (dimension1_beta_cellfactor2*dimension2_beta_cellfactor2)*sizeof(float)));




	return;
}



void mem_gpu_release()
{
	checkCudaErrors(hipFree(d_X_batch));
	checkCudaErrors(hipFree(d_Z_batch));
	checkCudaErrors(hipFree(d_Y_batch));
	checkCudaErrors(hipFree(d_Y_batch_exp));
	checkCudaErrors(hipFree(d_error_batch));
	checkCudaErrors(hipFree(d_cellfactor_batch));
	checkCudaErrors(hipFree(d_cellfactor_batch_new));

	checkCudaErrors(hipFree(d_list_cis_start));
	checkCudaErrors(hipFree(d_list_cis_end));
	checkCudaErrors(hipFree(d_list_beta_cis_start));
	checkCudaErrors(hipFree(d_list_beta_cis_geneindex));
	checkCudaErrors(hipFree(d_beta_cis_sub));
	checkCudaErrors(hipFree(d_der_cis_sub));

	checkCudaErrors(hipFree(d_beta_batch));
	checkCudaErrors(hipFree(d_beta_batch_reshape));
	checkCudaErrors(hipFree(d_der_batch));

	checkCudaErrors(hipFree(d_beta_cellfactor1));
	checkCudaErrors(hipFree(d_beta_cellfactor1_reshape));
	checkCudaErrors(hipFree(d_der_cellfactor1));

	checkCudaErrors(hipFree(d_beta_cellfactor2_sub));
	checkCudaErrors(hipFree(d_beta_cellfactor2_sub_reshape));
	checkCudaErrors(hipFree(d_der_cellfactor2_sub));


	return;
}





// init tissue data and parameters in tissue#k
void mem_gpu_settissue(int k)
{

	{// training
		//===============================================================
		//==== init and transmit tissue data when starting a new tissue
		//===============================================================
		int dimension1 = Y.get_dimension2_at(k);
		int dimension2 = J;
		int * list_indiv_pos = Y.get_list_indiv_pos_at(k);
		int dimension;

		//==== construct the genotype matrix for this tissue (with intercept term), on CPU then GPU memory
		//== X_sub, d_X_sub
		Matrix X_sub;
		dimension = X.get_dimension2();
		X_sub.init(dimension1, dimension);
		float * X_pointer = X.get_pointer();
		X_sub.fill_with_ref_list(list_indiv_pos, X_pointer);
		//
		checkCudaErrors(hipMalloc((void **) &d_X_sub, (dimension1*dimension)*sizeof(float)));
		float * X_sub_pointer = X_sub.get_pointer();
		checkCudaErrors(hipMemcpy(d_X_sub, X_sub_pointer, (dimension1*dimension)*sizeof(float), hipMemcpyHostToDevice));

		//== Z_sub, d_Z_sub
		Matrix Z_sub;
		dimension = Z.get_dimension2();
		Z_sub.init(dimension1, dimension);
		float * Z_pointer = Z.get_pointer();
		Z_sub.fill_with_ref_list(list_indiv_pos, Z_pointer);
		//
		checkCudaErrors(hipMalloc((void **) &d_Z_sub, (dimension1*dimension)*sizeof(float)));
		float * Z_sub_pointer = Z_sub.get_pointer();
		checkCudaErrors(hipMemcpy(d_Z_sub, Z_sub_pointer, (dimension1*dimension)*sizeof(float), hipMemcpyHostToDevice));

		//== d_Y_sub, d_Y_sub_exp
		float * Y_sub_pointer = Y.get_matrix_at(k);
		checkCudaErrors(hipMalloc((void **) &d_Y_sub, (dimension1*dimension2)*sizeof(float)));
		checkCudaErrors(hipMemcpy(d_Y_sub, Y_sub_pointer, (dimension1*dimension2)*sizeof(float), hipMemcpyHostToDevice));
		//
		checkCudaErrors(hipMalloc((void **) &d_Y_sub_exp, (dimension1*dimension2)*sizeof(float)));

		//== d_cellfactor_sub, d_cellfactor_sub_new
		checkCudaErrors(hipMalloc((void **) &d_cellfactor_sub, (dimension1*D)*sizeof(float)));
		//
		checkCudaErrors(hipMalloc((void **) &d_cellfactor_sub_new, (dimension1*(D+1))*sizeof(float)));


		//==##== collector ==##==
		X_sub.release();
		Z_sub.release();
	}


	//@@@@@@@@########@@@@@@@@
	// we have the testing set
	if(indicator_crossv)
	{
		int dimension1 = Y_test.get_dimension2_at(k);
		int dimension2 = J;
		int * list_indiv_pos = Y_test.get_list_indiv_pos_at(k);
		int dimension;

		//==== construct the genotype matrix for this tissue (with intercept term), on CPU then GPU memory
		//== X_subtest, d_X_subtest
		Matrix X_subtest;
		dimension = X_test.get_dimension2();
		X_subtest.init(dimension1, dimension);
		float * X_test_pointer = X_test.get_pointer();
		X_subtest.fill_with_ref_list(list_indiv_pos, X_test_pointer);
		//
		checkCudaErrors(hipMalloc((void **) &d_X_subtest, (dimension1*dimension)*sizeof(float)));
		float * X_subtest_pointer = X_subtest.get_pointer();
		checkCudaErrors(hipMemcpy(d_X_subtest, X_subtest_pointer, (dimension1*dimension)*sizeof(float), hipMemcpyHostToDevice));

		//== Z_subtest, d_Z_subtest
		Matrix Z_subtest;
		dimension = Z_test.get_dimension2();
		Z_subtest.init(dimension1, dimension);
		float * Z_test_pointer = Z_test.get_pointer();
		Z_subtest.fill_with_ref_list(list_indiv_pos, Z_test_pointer);
		//
		checkCudaErrors(hipMalloc((void **) &d_Z_subtest, (dimension1*dimension)*sizeof(float)));
		float * Z_subtest_pointer = Z_subtest.get_pointer();
		checkCudaErrors(hipMemcpy(d_Z_subtest, Z_subtest_pointer, (dimension1*dimension)*sizeof(float), hipMemcpyHostToDevice));

		//== d_Y_subtest, d_Y_subtest_exp
		float * Y_subtest_pointer = Y_test.get_matrix_at(k);
		checkCudaErrors(hipMalloc((void **) &d_Y_subtest, (dimension1*dimension2)*sizeof(float)));
		checkCudaErrors(hipMemcpy(d_Y_subtest, Y_subtest_pointer, (dimension1*dimension2)*sizeof(float), hipMemcpyHostToDevice));
		//
		checkCudaErrors(hipMalloc((void **) &d_Y_subtest_exp, (dimension1*dimension2)*sizeof(float)));

		//== d_cellfactor_subtest, d_cellfactor_subtest_new
		checkCudaErrors(hipMalloc((void **) &d_cellfactor_subtest, (dimension1*D)*sizeof(float)));
		//
		checkCudaErrors(hipMalloc((void **) &d_cellfactor_subtest_new, (dimension1*(D+1))*sizeof(float)));


		//==##== collector ==##==
		X_subtest.release();
		Z_subtest.release();
	}



	//===============================================================
	//==== transmit tissue parameters when starting a new tissue
	//===============================================================
	//== d_beta_cis_sub
	float * beta_cis_sub = beta_cis.get_incomp_matrix_at(k);
	int amount = beta_cis.get_amount();
	checkCudaErrors(hipMemcpy(d_beta_cis_sub, beta_cis_sub, amount*sizeof(float), hipMemcpyHostToDevice));

	//== d_beta_batch
	float * beta_batch_pointer = beta_batch.get_pointer();
	int dimension1_beta_batch = beta_batch.get_dimension1();
	int dimension2_beta_batch = beta_batch.get_dimension2();
	checkCudaErrors(hipMemcpy(d_beta_batch, beta_batch_pointer, (dimension1_beta_batch*dimension2_beta_batch)*sizeof(float), hipMemcpyHostToDevice));

	//== d_beta_cellfactor1
	float * beta_cellfactor1_pointer = beta_cellfactor1.get_pointer();
	int dimension1_beta_cellfactor1 = beta_cellfactor1.get_dimension1();
	int dimension2_beta_cellfactor1 = beta_cellfactor1.get_dimension2();
	checkCudaErrors(hipMemcpy(d_beta_cellfactor1, beta_cellfactor1_pointer, (dimension1_beta_cellfactor1*dimension2_beta_cellfactor1)*sizeof(float), hipMemcpyHostToDevice));

	//== d_beta_cellfactor2_sub
	float * beta_cellfactor2_pointer = beta_cellfactor2.get_matrix_at(k);
	int dimension1_beta_cellfactor2 = beta_cellfactor2.get_dimension2();
	int dimension2_beta_cellfactor2 = beta_cellfactor2.get_dimension3();
	checkCudaErrors(hipMemcpy(d_beta_cellfactor2_sub, beta_cellfactor2_pointer, (dimension1_beta_cellfactor2*dimension2_beta_cellfactor2)*sizeof(float), hipMemcpyHostToDevice));
	

	return;
}




// release tissue relevant containers, and copy back learned parameters for this tissue
void mem_gpu_destroytissue(int k)
{

	//==== release
	checkCudaErrors(hipFree(d_X_sub));
	checkCudaErrors(hipFree(d_Z_sub));
	checkCudaErrors(hipFree(d_Y_sub));
	checkCudaErrors(hipFree(d_Y_sub_exp));

	checkCudaErrors(hipFree(d_cellfactor_sub));
	checkCudaErrors(hipFree(d_cellfactor_sub_new));


	//@@@@@@@@########@@@@@@@@
	// we have the testing set
	if(indicator_crossv)
	{
		checkCudaErrors(hipFree(d_X_subtest));
		checkCudaErrors(hipFree(d_Z_subtest));
		checkCudaErrors(hipFree(d_Y_subtest));
		checkCudaErrors(hipFree(d_Y_subtest_exp));

		checkCudaErrors(hipFree(d_cellfactor_subtest));
		checkCudaErrors(hipFree(d_cellfactor_subtest_new));
	}


	//==== copy back parameters
	//== d_beta_cis_sub
	float * beta_cis_sub = beta_cis.get_incomp_matrix_at(k);
	int amount = beta_cis.get_amount();
	checkCudaErrors(hipMemcpy(beta_cis_sub, d_beta_cis_sub, amount*sizeof(float), hipMemcpyDeviceToHost));

	//== d_beta_batch
	float * beta_batch_pointer = beta_batch.get_pointer();
	int dimension1_beta_batch = beta_batch.get_dimension1();
	int dimension2_beta_batch = beta_batch.get_dimension2();
	checkCudaErrors(hipMemcpy(beta_batch_pointer, d_beta_batch, (dimension1_beta_batch*dimension2_beta_batch)*sizeof(float), hipMemcpyDeviceToHost));

	//== d_beta_cellfactor1
	float * beta_cellfactor1_pointer = beta_cellfactor1.get_pointer();
	int dimension1_beta_cellfactor1 = beta_cellfactor1.get_dimension1();
	int dimension2_beta_cellfactor1 = beta_cellfactor1.get_dimension2();
	checkCudaErrors(hipMemcpy(beta_cellfactor1_pointer, d_beta_cellfactor1, (dimension1_beta_cellfactor1*dimension2_beta_cellfactor1)*sizeof(float), hipMemcpyDeviceToHost));

	//== d_beta_cellfactor2_sub
	float * beta_cellfactor2_pointer = beta_cellfactor2.get_matrix_at(k);
	int dimension1_beta_cellfactor2 = beta_cellfactor2.get_dimension2();
	int dimension2_beta_cellfactor2 = beta_cellfactor2.get_dimension3();
	checkCudaErrors(hipMemcpy(beta_cellfactor2_pointer, d_beta_cellfactor2_sub, (dimension1_beta_cellfactor2*dimension2_beta_cellfactor2)*sizeof(float), hipMemcpyDeviceToHost));


	return;
}




