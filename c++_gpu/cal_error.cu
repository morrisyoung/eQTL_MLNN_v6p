#include "hip/hip_runtime.h"
// Utilities and system includes
#include <assert.h>
#include <helper_string.h>  // helper for shared functions common to CUDA Samples
#include <sys/time.h>
#include <time.h>       /* clock_t, clock, CLOCKS_PER_SEC */
#include <random>
#include <chrono>		/* sys time */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <vector>


// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// private
#include "cal_error.h"
#include "global.h"
#include "library.h"
#include "utility_gpu.cuh"




using namespace std;




// calculate the total squared error for specified tissue
float cal_error(int k)
{
	float error = 0;
	int dimension1 = Y.get_dimension2_at(k);
	int dimension2 = J;


	//==========================
	// from cis- (tissue k)
	//==========================
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==
	{
		int block_size = 32;
		dim3 threads(block_size, block_size);
		dim3 grid( (dimension2+threads.x-1)/threads.x, (dimension1+threads.y-1)/threads.y );
		kernel_cal_cis_matrixmul<32><<< grid, threads >>>(d_Y_sub_exp, dimension1, dimension2, d_X_sub, X.get_dimension2(), d_list_cis_start, d_list_cis_end, d_beta_cis_sub, d_list_beta_cis_start);
	}
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==




	//==========================
	// from batch
	//==========================
	int dimension1_beta_batch = beta_batch.get_dimension1();
	int dimension2_beta_batch = beta_batch.get_dimension2();
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==
	{
		int block_size = 32;
		dim3 threads(block_size, block_size);
		dim3 grid( (dimension2_beta_batch+threads.x-1)/threads.x, (dimension1_beta_batch+threads.y-1)/threads.y );
		kernel_op_matrix_reshape<32><<< grid, threads >>>(dimension1_beta_batch, dimension2_beta_batch, d_beta_batch, d_beta_batch_reshape);
	}
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==

	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==
	// func: d_Y_sub_exp += d_Z_sub x d_beta_batch_reshape;
	// dimension: (dimension1, dimension2) += (dimension1, dimension2_beta_batch) x (dimension2_beta_batch, dimension2)
	{
		const float alpha = 1.0f;
		const float beta  = 1.0f;									// NOTE: we always do cis- first with customer kernel, and then add on expression from batch and cell factors
		hipblasHandle_t handle;
		checkCudaErrors(hipblasCreate(&handle));
		//note cublas is column primary! need to transpose the order
		//checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));
		checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimension2, dimension1, dimension2_beta_batch, &alpha, d_beta_batch_reshape, dimension2, d_Z_sub, dimension2_beta_batch, &beta, d_Y_sub_exp, dimension2));
		checkCudaErrors(hipblasDestroy(handle));
	}
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==





	//=============
	// from cell factor (tissue k)
	//=============
	int dimension1_beta_cellfactor1 = beta_cellfactor1.get_dimension1();
	int dimension2_beta_cellfactor1 = beta_cellfactor1.get_dimension2();
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==
	{
		int block_size = 32;
		dim3 threads(block_size, block_size);
		dim3 grid( (dimension2_beta_cellfactor1+threads.x-1)/threads.x, (dimension1_beta_cellfactor1+threads.y-1)/threads.y );
		kernel_op_matrix_reshape<32><<< grid, threads >>>(dimension1_beta_cellfactor1, dimension2_beta_cellfactor1, d_beta_cellfactor1, d_beta_cellfactor1_reshape);
	}
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==

	// first layer
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==
	// func: d_cellfactor_sub = d_X_sub x d_beta_cellfactor1_reshape;
	// dimension: (dimension1, dimension1_beta_cellfactor1) += (dimension1, dimension2_beta_cellfactor1) x (dimension2_beta_cellfactor1, dimension1_beta_cellfactor1)
	{
		const float alpha = 1.0f;
		const float beta  = 0.0f;									// NOTE: add, other than add-on
		hipblasHandle_t handle;
		checkCudaErrors(hipblasCreate(&handle));
		//note cublas is column primary! need to transpose the order
		//checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));
		checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimension1_beta_cellfactor1, dimension1, dimension2_beta_cellfactor1, &alpha, d_beta_cellfactor1_reshape, dimension1_beta_cellfactor1, d_X_sub, dimension2_beta_cellfactor1, &beta, d_cellfactor_sub, dimension1_beta_cellfactor1));
		checkCudaErrors(hipblasDestroy(handle));
	}
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==

	// logistic twist: d_cellfactor_sub
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==
	{
		int block_size = 32;
		dim3 threads(block_size, block_size);
		dim3 grid( (dimension1_beta_cellfactor1+threads.x-1)/threads.x, (dimension1+threads.y-1)/threads.y );
		kernel_cal_matrix_logistic<32><<< grid, threads >>>(dimension1, dimension1_beta_cellfactor1, d_cellfactor_sub);
	}
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==

	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==
	{
		int block_size = 32;
		dim3 threads(block_size, block_size);
		dim3 grid( ((dimension1_beta_cellfactor1+1)+threads.x-1)/threads.x, (dimension1+threads.y-1)/threads.y );
		kernel_op_matrix_appendone<32><<< grid, threads >>>(dimension1, dimension1_beta_cellfactor1+1, d_cellfactor_sub_new, d_cellfactor_sub);
	}
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==

	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==
	int dimension1_beta_cellfactor2 = beta_cellfactor2.get_dimension2();
	int dimension2_beta_cellfactor2 = beta_cellfactor2.get_dimension3();
	{
		int block_size = 32;
		dim3 threads(block_size, block_size);
		dim3 grid( (dimension2_beta_cellfactor2+threads.x-1)/threads.x, (dimension1_beta_cellfactor2+threads.y-1)/threads.y );
		kernel_op_matrix_reshape<32><<< grid, threads >>>(dimension1_beta_cellfactor2, dimension2_beta_cellfactor2, d_beta_cellfactor2_sub, d_beta_cellfactor2_sub_reshape);
	}
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==

	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==
	// func: d_Y_sub_exp += d_cellfactor_sub_new x d_beta_cellfactor2_sub_reshape;
	// dimension: (dimension1, dimension2) += (dimension1, dimension2_beta_cellfactor2) x (dimension2_beta_cellfactor2, dimension2)
	{
		const float alpha = 1.0f;
		const float beta  = 1.0f;									// NOTE: add-on
		hipblasHandle_t handle;
		checkCudaErrors(hipblasCreate(&handle));
		//note cublas is column primary! need to transpose the order
		//checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));
		checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimension2, dimension1, dimension2_beta_cellfactor2, &alpha, d_beta_cellfactor2_sub_reshape, dimension2, d_cellfactor_sub_new, dimension2_beta_cellfactor2, &beta, d_Y_sub_exp, dimension2));
		checkCudaErrors(hipblasDestroy(handle));
	}
	//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==//==






	//=============
	// compile and error cal
	//=============
	{
		// two steps: sub sum; sum sub
		//== d_sumY_temp, d_sum
		float * d_sumY_temp;
		int sub_amount = 200;									// TODO: to tune this number
		int sub_length = (dimension1*dimension2 + sub_amount-1) / sub_amount;
		checkCudaErrors(hipMalloc((void **) &d_sumY_temp, sub_length*sizeof(float)));
		//
		float * d_sum;
		checkCudaErrors(hipMalloc((void **) &d_sum, 1*sizeof(float)));
		float h_sum;

		int block_size = 32;
		dim3 threads(block_size);
		dim3 grid( (sub_length+threads.x-1)/threads.x );
		//
		kernel_cal_sosod_subsum<32><<< grid, threads >>>(sub_amount, sub_length, dimension1*dimension2, d_sumY_temp, d_Y_sub_exp, d_Y_sub);
		//
		kernel_cal_sosod_sumsub<32><<< grid, threads >>>(sub_length, d_sumY_temp, d_sum);
		//
		checkCudaErrors(hipMemcpy(h_sum, d_sum, 1*sizeof(float), hipMemcpyDeviceToHost));
		error = h_sum;

		//==##== collector ==##==
		checkCudaErrors(hipFree(d_sumY_temp));
		checkCudaErrors(hipFree(d_sum));
	}





	return error;
}



